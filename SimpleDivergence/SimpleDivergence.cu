#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

inline double seconds()
{
        struct timeval tp;
	struct timezone tzp;
        int i = gettimeofday(&tp, &tzp);
	return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
	//return ((double)tp.tv_sec*1e6 + (double)tp.tv_usec );
}


__global__ void mathkernel1(float *c)
{
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	float a,b;
	a=b=0.0f;

	if(tid%2==0)
	{
		a=100.0;
	}
	else
	{
		b=200.0;
	}
	c[tid]=a+b;
//        printf("c[%d]= %f\n",tid,c[tid]);
}

__global__ void mathkernel2(float *c)
{
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	float a,b;
	a=b=0.0f;

	if((tid/warpSize)%2==0)
	{
		a=100.0;
	}
	else
	{
		b=200.0;
	}
	c[tid]=a+b;
        //printf("c[%d]= %f\n",tid,c[tid]);
}
__global__ void warmingup(float *c)
{
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	float a,b;
	a=b=0.0f;

	if((tid/warpSize)%2==0)
	{
		a=100.0;
	}
	else
	{
		b=200.0;
	}
	c[tid]=a+b;
}




int main(int argc, char **argv)
{
	//set up device
	int dev=0;
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp,dev);
	printf("%s using Device %d: %s\n",argv[0],dev,deviceProp.name);

	//set up data size
	int size=64;
	int blocksize=64;
	if(argc>1)blocksize=atoi(argv[1]);
	if(argc>2)size=atoi(argv[2]);
	printf("Data size %d\n ",size);

	//set up execution configuration
	dim3 block(blocksize,1);
	dim3 grid((size+block.x-1)/block.x,1);
	printf("Execution Configure (block %d grid %d)\n",block.x,grid.x);

	//allocate gpu memory
	float *d_C;
        size_t nBytes=size*sizeof(float);
	hipMalloc((float **)&d_C,nBytes);

	//run a warmup kernel to remove overhead
	double iStart,iElaps;
	hipDeviceSynchronize();
	iStart=seconds();
	warmingup<<<grid,block>>> (d_C);
	hipDeviceSynchronize();
	iElaps=seconds()-iStart;
	printf("warmup    <<<%4d %4d>>> elapsed %f sec \n",grid.x,block.x,iElaps);
	
	//run kernel 1
	iStart=seconds();
	mathkernel1<<<grid,block>>> (d_C);
	hipDeviceSynchronize();
	iElaps=seconds()-iStart;
	printf("mathkernel1<<<%4d %4d>>> elapsed %f sec \n",grid.x,block.x,iElaps);
	//run kernel 2
	iStart=seconds();
	mathkernel2<<<grid,block>>> (d_C);
	hipDeviceSynchronize();
	iElaps=seconds()-iStart;
	printf("mathkernel2<<<%4d %4d>>> elapsed %f sec \n",grid.x,block.x,iElaps);
	//run kernel 3
	iStart=seconds();
	//mathkernel3<<<grid,block>>> (d_C);
	hipDeviceSynchronize();
	iElaps=seconds()-iStart;
	printf("mathkernel3<<<%4d %4d>>> elapsed %f sec \n",grid.x,block.x,iElaps);
	//run kernel 4
	iStart=seconds();
	//mathkernel4<<<grid,block>>> (d_C);
	hipDeviceSynchronize();
	iElaps=seconds()-iStart;
	printf("mathkernel4<<<%4d %4d>>> elapsed %f sec \n",grid.x,block.x,iElaps);

	//free gpu memory and reset device
	hipFree(d_C);
	hipDeviceReset();
	return EXIT_SUCCESS;
}
